#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <chrono>
#include "calc_on_fpga.h"

// static const int numthread = 256;

// __global__
// void vecadd(float *a, float *b, float *c, int n) {
//   int i = blockIdx.x * blockDim.x + threadIdx.x;
//   int j = blockIdx.y * blockDim.y + threadIdx.y;
//   if (i < n || j < n) {
//     c[i*n+j] += a[i*n+k] * b[k*n+j];
//   }
// }

int main(int argc, char *argv[]) {
  // check command line arguments
  ///////////////////////////////////////////
  if (argc == 1) { std::cout << "usage: ./host <name> <numdata_h> <numstream> <numtry>" << std::endl; exit(0); }
  if (argc != 5) { std::cerr << "Error! The number of arguments is wrong."              << std::endl; exit(1); }

  const char *name     = argv[1];
  const int  numdata_h = std::stoull(std::string(argv[2]));
  const int  numstream = std::stoull(std::string(argv[3]));
  const int  numtry    = std::stoull(std::string(argv[4]));
  const int  numbyte   = numdata_h * sizeof(float); // this sample uses "float"
  const int  numdata_d = (numdata_h/numstream);

  size_t global_item_size[3];
  size_t local_item_size[3];
  
  // host memory settings
  ///////////////////////////////////////////

  /***** FPGA *****/
  static CalcOnFPGA calc_on_fpga;
  int N = 1000;
  int K = 1000;
  int VAL_SIZE = 1000;
  float *FPGA_calc_result; // = new float[N];
  float *VAL;
  int *COL_IND;
  int *ROW_PTR;
  float *B;

  posix_memalign((void **)&FPGA_calc_result, 64, N * sizeof(float));
  posix_memalign((void **)&VAL, 64, VAL_SIZE * sizeof(float));
  posix_memalign((void **)&COL_IND, 64, VAL_SIZE * sizeof(int));
  posix_memalign((void **)&ROW_PTR, 64, N+1 * sizeof(int));
  posix_memalign((void **)&B, 64, N * sizeof(float));

  for(int i=0; i<VAL_SIZE; i++) {
    VAL[i] = i+1.0f;
    COL_IND[i] = i;
  }
  for(int j=0; j<N; j++) {
    // FPGA_calc_result[j] = 0;
    ROW_PTR[j] = j;
    B[j] = j/2 - 0.0f; // x=0.0; b - Ax
  }
  ROW_PTR[N] = N;

  calc_on_fpga.InitOpenCL(name, N, K, VAL_SIZE, global_item_size, local_item_size);

  // main routine
  ///////////////////////////////////////////
  // const int numblock = (numdata_h % numthread) ? (numdata_h/numthread) + 1 : (numdata_h/numthread);
  
  std::chrono::system_clock::time_point start = std::chrono::system_clock::now();

  // clWaitForEvents(1, &calc_on_fpga.kernel_event);
  calc_on_fpga.SendDatatoFPGA(N, VAL_SIZE, VAL, COL_IND, ROW_PTR, B);
  calc_on_fpga.Exec(global_item_size, local_item_size);  // kernel running
  // getting the computation results
  calc_on_fpga.RecvDatafromFPGA(N, FPGA_calc_result);

  std::chrono::system_clock::time_point end = std::chrono::system_clock::now();
  
  std::cout << std::string(30, '-') << std::endl;
  std::cout << "elapsed time: " << std::fixed << std::chrono::duration_cast<std::chrono::seconds>(end-start).count() << " sec" << std::endl;

  // verification
  ///////////////////////////////////////////
  calc_on_fpga.Verify(FPGA_calc_result, VAL, COL_IND, ROW_PTR, B, N, K, VAL_SIZE);
    
  // cleanup
  ///////////////////////////////////////////
  // hipHostFree(h_a);
  // hipHostFree(h_b);
  // hipHostFree(h_c);
  // hipFree(d_a);
  // hipFree(d_b);
  // hipFree(d_c);
  // for (int stm = 0; stm < numstream; ++stm) {
  //   hipStreamDestroy(stream[stm]);
  // }

  delete[] FPGA_calc_result;
  
  return 0;
}
