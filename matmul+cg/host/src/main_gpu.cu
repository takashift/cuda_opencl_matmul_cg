
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <chrono>
// #include "calc_on_fpga.h"

__global__ void matmul(float *a, float *b, float *c, unsigned long n) {
  unsigned long j = blockIdx.x * blockDim.x + threadIdx.x; // 通し番号を得るための計算
  unsigned long i = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned long k;
  float sum = 0.0f;
  if (i < n || j < n) {
    for(k=0; k<n; k++) {
      sum += a[i*n+k] * b[k*n+j];
    }
    c[i*n+j] = sum;
  }
}

void MatrixMultiplication_openmp(float * a,float * b, float * c, unsigned long N)
{
  int i, j, k ;
  int chunk = N/4;


#pragma omp parallel shared(a,b,c,chunk) private(i,j,k)
  {
#ifdef _OPENMP
	if(omp_get_thread_num() == 0) {
		printf("Number of OpenMP threads %d\n", omp_get_num_threads());
	}
#endif
#pragma omp for
    for (i=0; i<N; i++){
      for (j=0; j<N; j++)
        {
	  float sum = 0.0 ;
	  for (k=0; k<N; k++)
	    sum += a[i*N+k]*b[k*N+j] ;
	  c[i*N+j] = sum ;
        }
    }
  }
}

int main(int argc, char *argv[]) {
  // check command line arguments
  ///////////////////////////////////////////
  if (argc == 1) { std::cout << "usage: ./host <name> <numdata_h> <numstream> <numtry>" << std::endl; exit(0); }
  if (argc != 5) { std::cerr << "Error! The number of arguments is wrong."              << std::endl; exit(1); }

  const char *name     = argv[1];
  const unsigned long numdata_h = std::stoull(std::string(argv[2]));
  const int  numstream = std::stoull(std::string(argv[3]));
  const int  numtry    = std::stoull(std::string(argv[4]));
  const unsigned long numbyte   = numdata_h * numdata_h * sizeof(float); // this sample uses "float"

  // size_t global_item_size[3];
  // size_t local_item_size[3];
  
  // host memory settings
  ///////////////////////////////////////////

  /***** GPU *****/
  static const int numthread = 256;  
  const unsigned long numblock = (numdata_h % numthread) ? (numdata_h/numthread) + 1 : (numdata_h/numthread);
  float *h_a, *h_b, *h_c, *c_CPU;

  hipHostMalloc(&h_a, numbyte, hipHostMallocDefault);
  hipHostMalloc(&h_b, numbyte, hipHostMallocDefault);
  hipHostMalloc(&h_c, numbyte, hipHostMallocDefault);
  
  for (unsigned long i = 0; i < numdata_h; i++) {
    for (unsigned long j = 0; j < numdata_h; j++) {
      h_a[i*numdata_h+j] = 0.0f; //(j+1)/2*0.0001f;
      h_b[i*numdata_h+j] = 0.5f;
      h_c[i*numdata_h+j] = 0.0f;
      c_CPU[i*numdata_h+j] = 0.0f;
    }
  }

  // device memory settings
  ///////////////////////////////////////////
  float *d_a, *d_b, *d_c;

  hipMalloc(&d_a, numbyte);
  hipMalloc(&d_b, numbyte);
  hipMalloc(&d_c, numbyte);

  // main routine
  ///////////////////////////////////////////
  
  std::chrono::system_clock::time_point start = std::chrono::system_clock::now();

  hipMemcpy(d_a, h_a, numbyte, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, numbyte, hipMemcpyHostToDevice);
  
  matmul<<<numblock, numthread>>>(d_a, d_b, d_c, numdata_h);
  
  hipMemcpy(h_c, d_c, numbyte, hipMemcpyDeviceToHost);

  std::chrono::system_clock::time_point end = std::chrono::system_clock::now();
  
  // verification
  ///////////////////////////////////////////
  MatrixMultiplication_openmp(h_a, h_b, c_CPU, numdata_h);
  double cpu_sum = 0.0;
  double gpu_sum = 0.0;
  double rel_err = 0.0;

  for (int i=0; i<numdata_h*numdata_h; i++){
    cpu_sum += (double)c_CPU[i]*c_CPU[i];
    gpu_sum += (double)h_c[i]*h_c[i];
  }

  cpu_sum = sqrt(cpu_sum);
  gpu_sum = sqrt(gpu_sum);
  if( cpu_sum > gpu_sum ) {
    rel_err = (cpu_sum-gpu_sum)/cpu_sum;
  } else {
    rel_err = (gpu_sum-cpu_sum)/cpu_sum;
  }

  if(rel_err < 1e-6)
  {
      printf("Verification Successful err = %e\n", rel_err);
  }
  else
  {
      printf("Verification Fail err = %e\n", rel_err);
  }
  printf("ResultGPU = %lf\n", gpu_sum);
  printf("ResultCPU = %lf\n", cpu_sum);

    std::cout << std::string(30, '-') << std::endl;
    std::cout << "elapsed time: " << std::fixed << std::chrono::duration_cast<std::chrono::microseconds>(end-start).count() << " usec" << std::endl;
    
  // cleanup
  ///////////////////////////////////////////
  hipHostFree(h_a);
  hipHostFree(h_b);
  hipHostFree(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // delete[] FPGA_calc_result;
  
  return 0;
}
