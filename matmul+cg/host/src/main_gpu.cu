#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <chrono>
#include "calc_on_fpga.h"

static const int numthread = 256;

__global__ void matmul(float *a, float *b, float *c, int n) {
  int j = blockIdx.x * blockDim.x + threadIdx.x; // 通し番号を得るための計算
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int k;
  float sum = 0.0f;
  if (i < n || j < n) {
    for(k =0; k<n; k++) {
      sum += a[i*n+k] * b[k*n+j];
    }
    c[i*n+j] = sum;
  }
}

int main(int argc, char *argv[]) {
  // check command line arguments
  ///////////////////////////////////////////
  if (argc == 1) { std::cout << "usage: ./host <name> <numdata_h> <numstream> <numtry>" << std::endl; exit(0); }
  if (argc != 5) { std::cerr << "Error! The number of arguments is wrong."              << std::endl; exit(1); }

  const char *name     = argv[1];
  const int  numdata_h = std::stoull(std::string(argv[2]));
  const int  numstream = std::stoull(std::string(argv[3]));
  const int  numtry    = std::stoull(std::string(argv[4]));
  const int  numbyte   = numdata_h * numdata_h * sizeof(float); // this sample uses "float"
  const int  numdata_d = (numdata_h/numstream);

  size_t global_item_size[3];
  size_t local_item_size[3];
  
  // host memory settings
  ///////////////////////////////////////////

  /***** GPU *****/
  float *h_a, *h_b, *h_c;
  dim3 block(64, 64), thread(16, 16);

  hipHostMalloc(&h_a, numbyte);
  hipHostMalloc(&h_b, numbyte);
  hipHostMalloc(&h_c, numbyte);
  
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      h_a[i*N+j] = (j+1)*0.0001f;
      h_b[i*N+j] = 0.5f;
      h_c[i*N+j] = 0.0f;
    }
  }

  // device memory settings
  ///////////////////////////////////////////
  float *d_a, *d_b, *d_c;

  hipMalloc(&d_a, numbyte);
  hipMalloc(&d_b, numbyte);
  hipMalloc(&d_c, numbyte);

  // main routine
  ///////////////////////////////////////////
  
  std::chrono::system_clock::time_point start = std::chrono::system_clock::now();

  hipMemcpy(d_a, h_a, numbyte, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, numbyte, hipMemcpyHostToDevice);
  
  matmul<<<block, thread>>>(d_a, d_b, d_c, numdata_h);
  
  hipMemcpy(h_c, d_c, numbyte, hipMemcpyDeviceToHost);

  std::chrono::system_clock::time_point end = std::chrono::system_clock::now();
  
  // verification
  ///////////////////////////////////////////


    std::cout << std::string(30, '-') << std::endl;
    std::cout << "elapsed time: " << std::fixed << std::chrono::duration_cast<std::chrono::seconds>(end-start).count() << " sec" << std::endl;
    
  // cleanup
  ///////////////////////////////////////////
  hipHostFree(h_a);
  hipHostFree(h_b);
  hipHostFree(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  delete[] FPGA_calc_result;
  
  return 0;
}
